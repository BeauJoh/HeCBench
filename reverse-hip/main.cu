#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__global__ void reverse (int* d, const int len)
{
  __shared__ int s[256];
  int t = threadIdx.x;
  int tr = len-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main() {
  const int len = 256;
  const int iteration = 1 << 20;
  int d[len];
  for (int i = 0; i < len; i++) d[i] = i;

  int *dd;
  hipMalloc((void**)&dd, sizeof(int)*len);
  hipMemcpy(dd, d, sizeof(int)*len, hipMemcpyHostToDevice);
  for (int i = 0; i <= iteration; i++)
    hipLaunchKernelGGL(reverse, dim3(1), dim3(256), 0, 0, dd, len);
  hipMemcpy(d, dd, sizeof(int)*len, hipMemcpyDeviceToHost);
  hipFree(dd);

  for (int i = 0; i < len; i++) assert(d[i] == len-i-1);
  printf("PASS\n");

  return 0;
}
