#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <ctime>
#include <cstdio>
#include "device_kernel_wrapper.h"
#include "datatypes.h"
#include "common.h"
#include "memory_scheduler.h"


__global__
void device_chain_tiled(
        return_dt *ret, const anchor_dt *a,
        const control_dt *control, score_dt **max_tracker, parent_dt **j_tracker,
        const int max_dist_x, const int max_dist_y, const int bw);

__host__
void device_chain_kernel_wrapper(
        std::vector<control_dt> &cont,
        std::vector<anchor_dt> &arg,
        std::vector<return_dt> &ret,
        int max_dist_x, int max_dist_y, int bw)
{
    auto batch_count = cont.size() / PE_NUM;

    control_dt *h_control;
    anchor_dt *h_arg;
    return_dt *h_ret;

    hipHostMalloc(&h_control, cont.size() * sizeof(control_dt));
    hipHostMalloc(&h_arg, arg.size() * sizeof(anchor_dt));
    hipHostMalloc(&h_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
    ret.resize(batch_count * TILE_SIZE * PE_NUM);

    memcpy(h_control, cont.data(), cont.size() * sizeof(control_dt));
    memcpy(h_arg, arg.data(), arg.size() * sizeof(anchor_dt));

    struct timespec start, end;
    clock_gettime(CLOCK_BOOTTIME, &start);

    control_dt *d_control;
    anchor_dt *d_arg;
    return_dt *d_ret;

    // presistent storage
    score_dt *d_max_tracker[PE_NUM];
    parent_dt *d_j_tracker[PE_NUM];

    score_dt **d_d_max_tracker;
    parent_dt **d_d_j_tracker;

    hipMalloc(&d_control, cont.size() * sizeof(control_dt));
    hipMalloc(&d_arg, arg.size() * sizeof(anchor_dt));
    hipMalloc(&d_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));

    for (auto pe = 0; pe < PE_NUM; pe++) {
        hipMalloc(&d_max_tracker[pe], BACK_SEARCH_COUNT_GPU * sizeof(score_dt));
        hipMalloc(&d_j_tracker[pe], BACK_SEARCH_COUNT_GPU * sizeof(parent_dt));
    }
    hipMalloc(&d_d_max_tracker, PE_NUM * sizeof(score_dt *));
    hipMalloc(&d_d_j_tracker, PE_NUM * sizeof(parent_dt *));

    hipMemcpy(d_control, h_control,
            cont.size() * sizeof(control_dt), hipMemcpyHostToDevice);
    hipMemcpy(d_arg, h_arg,
            arg.size() * sizeof(anchor_dt), hipMemcpyHostToDevice);
    hipMemcpy(d_d_max_tracker, d_max_tracker,
            PE_NUM * sizeof(score_dt *), hipMemcpyHostToDevice);
    hipMemcpy(d_d_j_tracker, d_j_tracker,
            PE_NUM * sizeof(parent_dt *), hipMemcpyHostToDevice);


    for (auto batch = 0; batch < batch_count; batch++) {
            device_chain_tiled<<<BLOCK_NUM, BACK_SEARCH_COUNT_GPU>>> (
                    d_ret + batch * PE_NUM * TILE_SIZE,
                    d_arg + batch * PE_NUM * TILE_SIZE_ACTUAL,
                    d_control + batch * PE_NUM ,
                    d_d_max_tracker,
                    d_d_j_tracker,
                    max_dist_x, max_dist_y, bw);
    }

    hipMemcpy(h_ret, d_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt), hipMemcpyDeviceToHost);

    hipFree(d_control);
    hipFree(d_arg);
    hipFree(d_ret);
    for (auto pe = 0; pe < PE_NUM; pe++) {
        hipFree(d_max_tracker[pe]);
        hipFree(d_j_tracker[pe]);
    }
    hipFree(d_d_max_tracker);
    hipFree(d_d_j_tracker);

    clock_gettime(CLOCK_BOOTTIME, &end);
    printf(" ***** offloading took %f seconds for end-to-end\n",
        ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / 1E9);

    memcpy(ret.data(), h_ret, batch_count * TILE_SIZE * PE_NUM * sizeof(return_dt));
}

