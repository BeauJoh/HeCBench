#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__global__ void reverse (int* d, const int len)
{
  __shared__ int s[256];
  int t = threadIdx.x;
  int tr = len-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main() {
  const int len = 256;
  const int iteration = 1 << 20;
  int d[len];
  for (int i = 0; i < len; i++) d[i] = i;

  int *dd;
  hipMalloc((void**)&dd, sizeof(int)*len);
  hipMemcpy(dd, d, sizeof(int)*len, hipMemcpyHostToDevice);
  for (int i = 0; i <= iteration; i++)
    reverse<<<1, 256>>> (dd, len);
  hipMemcpy(d, dd, sizeof(int)*len, hipMemcpyDeviceToHost);
  hipFree(dd);

  for (int i = 0; i < len; i++) assert(d[i] == len-i-1);
  printf("PASS\n");

  return 0;
}
