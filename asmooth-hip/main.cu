#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "reference.cpp"

__global__ void smoothingFilter(
    int Lx, int Ly, 
    int Threshold, int MaxRad, 
    const float*__restrict Img,
            int*__restrict Box,
          float*__restrict Norm)
{
  int tid = threadIdx.x;
  int tjd = threadIdx.y;
  int i = blockIdx.x * blockDim.x + tid;
  int j = blockIdx.y * blockDim.y + tjd;
  int stid = tjd * blockDim.x + tid;
  int gtid = j * Lx + i;  

  // part of shared memory may be unused
  __shared__ float s_Img[1024];

  if ( i < Lx && j < Ly )
    s_Img[stid] = Img[gtid];

  __syncthreads();

  if ( i < Lx && j < Ly )
  {
    // Smoothing parameters
    float sum = 0.f;
    int q = 1;
    int s = q;
    int ksum = 0;

    // Continue until parameters are met
    while (sum < Threshold && q < MaxRad)
    {
      s = q;
      sum = 0.f;
      ksum = 0;

      // Normal adaptive smoothing
      for (int ii = -s; ii < s+1; ii++)
        for (int jj = -s; jj < s+1; jj++)
          if ( (i-s >= 0) && (i+s < Ly) && (j-s >= 0) && (j+s < Lx) )
          {
            ksum++;
            // Compute within bounds of block dimensions
            if( tid-s >= 0 && tid+s < blockDim.x && tjd-s >= 0 && tjd+s < blockDim.y )
              sum += s_Img[stid + ii*blockDim.x + jj];
            // Compute block borders with global memory
            else
              sum += Img[gtid + ii*Lx + jj];
          }
      q++;
    }
    Box[gtid] = s;

    // Normalization for each box
    for (int ii = -s; ii < s+1; ii++)
      for (int jj = -s; jj < s+1; jj++)
        if (ksum != 0) 
          atomicAdd(&Norm[gtid + ii*Lx + jj], __fdividef(1.f, (float)ksum));
  }
}

__global__ void normalizeFilter(int Lx, int Ly, float*__restrict Img, const float*__restrict Norm)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if ( i < Lx && j < Ly ) {
    int gtid = j * Lx + i;  
    const float norm = Norm[gtid];
    if (norm != 0) Img[gtid] = __fdividef(Img[gtid], norm);
  }
}

__global__ void outFilter( 
    int Lx, int Ly,
    const float*__restrict Img,
    const   int*__restrict Box,
          float*__restrict Out )
{
  int tid = threadIdx.x;
  int tjd = threadIdx.y;
  int i = blockIdx.x * blockDim.x + tid;
  int j = blockIdx.y * blockDim.y + tjd;
  int stid = tjd * blockDim.x + tid;
  int gtid = j * Lx + i;  

  // part of shared memory may be unused
  __shared__ float s_Img[1024];

  if ( i < Lx && j < Ly )
    s_Img[stid] = Img[gtid];

  __syncthreads();

  if ( i < Lx && j < Ly )
  {
    const int s = Box[gtid];
    float sum = 0.f;
    int ksum  = 0;

    for (int ii = -s; ii < s+1; ii++)
      for (int jj = -s; jj < s+1; jj++)
        if ( (i-s >= 0) && (i+s < Lx) && (j-s >= 0) && (j+s < Ly) )
        {
          ksum++;
          if( tid-s >= 0 && tid+s < blockDim.x && tjd-s >= 0 && tjd+s < blockDim.y )
            sum += s_Img[stid + ii*blockDim.y + jj];
          else
            sum += Img[gtid + ii*Ly + jj];
        }
    if ( ksum != 0 ) Out[gtid] = __fdividef(sum , (float)ksum);
  }
}

int main(int argc, char* argv[]) {
  if (argc != 5) {
     printf("./%s <image dimension> <threshold> <max box size> <iterations>\n", argv[0]);
     exit(1);
  }

  // only a square image is supported
  const int Lx = atoi(argv[1]);
  const int Ly = Lx;

  const int Threshold = atoi(argv[2]);
  const int MaxRad = atoi(argv[3]);
  const int repeat = atoi(argv[4]);
 
  // input image
  float *img = (float*) malloc (sizeof(float) * Lx * Ly);

  // host and device results
  float *norm = (float*) malloc (sizeof(float) * Lx * Ly);
  float *h_norm = (float*) malloc (sizeof(float) * Lx * Ly);

  int *box = (int*) malloc (sizeof(int) * Lx * Ly);
  int *h_box = (int*) malloc (sizeof(int) * Lx * Ly);

  float *out = (float*) malloc (sizeof(float) * Lx * Ly);
  float *h_out = (float*) malloc (sizeof(float) * Lx * Ly);

  srand(123);
  for (int i = 0; i < Lx * Ly; i++) {
    img[i] = rand() % 256;
    norm[i] = box[i] = out[i] = 0;
  }

  float *d_img;
  hipMalloc((void**)&d_img, sizeof(float) * Lx * Ly);

  float *d_norm;
  hipMalloc((void**)&d_norm, sizeof(float) * Lx * Ly);

  int *d_box;
  hipMalloc((void**)&d_box, sizeof(int) * Lx * Ly);

  float *d_out;
  hipMalloc((void**)&d_out, sizeof(float) * Lx * Ly);

  dim3 grids ((Lx+15)/16, (Ly+15)/16);
  dim3 blocks (16, 16);

  for (int i = 0; i < repeat; i++) {
    // restore input image
    hipMemcpy(d_img, img, sizeof(float) * Lx * Ly, hipMemcpyHostToDevice);
    // reset norm
    hipMemcpy(d_norm, norm, sizeof(float) * Lx * Ly, hipMemcpyHostToDevice);
    // launch three kernels
    hipLaunchKernelGGL(smoothingFilter, grids, blocks, 0, 0, Lx, Ly, Threshold, MaxRad, d_img, d_box, d_norm);
    hipLaunchKernelGGL(normalizeFilter, grids, blocks, 0, 0, Lx, Ly, d_img, d_norm);
    hipLaunchKernelGGL(outFilter, grids, blocks, 0, 0, Lx, Ly, d_img, d_box, d_out);
  }

  hipMemcpy(out, d_out, sizeof(float) * Lx * Ly, hipMemcpyDeviceToHost);
  hipMemcpy(box, d_box, sizeof(int) * Lx * Ly, hipMemcpyDeviceToHost);
  hipMemcpy(norm, d_norm, sizeof(float) * Lx * Ly, hipMemcpyDeviceToHost);

  // verify
  reference (Lx, Ly, Threshold, MaxRad, img, h_box, h_norm, h_out);

  bool ok = true;
  int cnt[10] = {0,0,0,0,0,0,0,0,0,0};
  for (int i = 0; i < Lx * Ly; i++) {
    if (fabsf(norm[i] - h_norm[i]) > 1e-3f) {
      printf("%d %f %f\n", i, norm[i], h_norm[i]);
      ok = false;
      break;
    }
    if (fabsf(out[i] - h_out[i]) > 1e-3f) {
      printf("%d %f %f\n", i, out[i], h_out[i]);
      ok = false;
      break;
    }
    if (box[i] != h_box[i]) {
      printf("%d %d %d\n", i, box[i], h_box[i]);
      ok = false;
      break;
    } else {
      for (int j = 0; j < MaxRad; j++)
        if (box[i] == j) { cnt[j]++; break; }
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");
  if (ok) {
    printf("Distribution of box sizes:\n");
    for (int j = 1; j < MaxRad; j++)
      printf("size=%d: %f\n", j, (float)cnt[j]/(Lx*Ly));
  }

  hipFree(d_img);
  hipFree(d_norm);
  hipFree(d_box);
  hipFree(d_out);
  free(img);
  free(norm);
  free(h_norm);
  free(box);
  free(h_box);
  free(out);
  free(h_out);
  return 0;
}
