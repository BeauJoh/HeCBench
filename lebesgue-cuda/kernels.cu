#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void kernel (double *__restrict__ lfun,
             double *__restrict__ linterp,
             const double *__restrict__ xfun, 
             const double *__restrict__ x,
             const int n, const int nfun) 
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j >= nfun) return;
  if ( n == 1 ) {
    lfun[j] = 1.0;
  } else {
    for (int i = 0; i < n; i++ )
      linterp[i*nfun+j] = 1.0;

    for (int i1 = 0; i1 < n; i1++ )
      for (int i2 = 0; i2 < n; i2++ )
        if ( i1 != i2 )
          linterp[i1*nfun+j] = linterp[i1*nfun+j] * ( xfun[j] - x[i2] ) / ( x[i1] - x[i2] );

    double t = 0.0;
    for (int i = 0; i < n; i++ )
      t += fabs ( linterp[i*nfun+j] );

    lfun[j] = t;
  }
}

double *lebesgue_function ( int n, double x[], int nfun, double xfun[] )
{
  double *lfun = ( double * ) malloc ( nfun * sizeof ( double ) );

  double *d_fun, *d_interp, *d_xfun, *d_x;
  hipMalloc((void**)&d_fun, nfun * sizeof ( double ) );
  hipMalloc((void**)&d_interp, n * nfun * sizeof ( double ) );
  hipMalloc((void**)&d_x, n * sizeof ( double ) );
  hipMalloc((void**)&d_xfun, nfun * sizeof ( double ) );

  hipMemcpy(d_x, x, n * sizeof ( double ), hipMemcpyHostToDevice );
  hipMemcpy(d_xfun, xfun, nfun * sizeof ( double ), hipMemcpyHostToDevice );
  
  dim3 grids ((nfun + 255)/256);
  dim3 blocks (256);

  kernel<<<grids, blocks>>> (d_fun, d_interp, d_xfun, d_x, n, nfun);
  hipMemcpy(lfun, d_fun, nfun * sizeof ( double ), hipMemcpyDeviceToHost );
  hipFree(d_fun);
  hipFree(d_interp);
  hipFree(d_xfun);
  hipFree(d_x);
  return lfun;
}
